/*
* BSD 3-Clause License
*
* Copyright (c) 2017-2018, plures
* All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions are met:
*
* 1. Redistributions of source code must retain the above copyright notice,
*    this list of conditions and the following disclaimer.
*
* 2. Redistributions in binary form must reproduce the above copyright notice,
*    this list of conditions and the following disclaimer in the documentation
*    and/or other materials provided with the distribution.
*
* 3. Neither the name of the copyright holder nor the names of its
*    contributors may be used to endorse or promote products derived from
*    this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
* AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
* DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
* FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
* DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
* SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
* CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
* OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/


#include <stdint.h>
#include <ndtypes.h>


/*****************************************************************************/
/*                  Expose some cuda memory functions                        */
/*****************************************************************************/

extern "C" void *
xnd_cuda_calloc_managed(uint16_t align, int64_t size, ndt_context_t *ctx)
{
    char *ptr;
    hipError_t err;

    size = size == 0 ? 1 : size;

    if (size < 0 || (uint64_t)size > SIZE_MAX) {
        ndt_err_format(ctx, NDT_ValueError,
            "hipMallocManaged: invalid size");
        return NULL;
    }

    err = hipMallocManaged(&ptr, (size_t)size);
    if (err != hipSuccess) {
        ndt_err_format(ctx, NDT_MemoryError,
            "hipMallocManaged: allocation failed");
        return NULL;
    }

    if (((uintptr_t)ptr) % align != 0) {
        ndt_err_format(ctx, NDT_ValueError,
            "hipMallocManaged: alignment requirement too large");
        hipFree(ptr);
        return NULL;
    }

    memset(ptr, '\0', (size_t)size);

    return ptr;
}

extern "C" void
xnd_cuda_free(void *ptr)
{
    hipError_t err;

    err = hipFree(ptr);
    if (err != hipSuccess) {
        fprintf(stderr, "hipFree: unexpected failure\n");
    }
}

extern "C" int
xnd_cuda_mem_prefetch_async(const void *ptr, int64_t count, int dev,
                            ndt_context_t *ctx)
{
    hipError_t err;

    if (count < 0 || (uint64_t)count > SIZE_MAX) {
        ndt_err_format(ctx, NDT_ValueError,
            "hipMemPrefetchAsync: invalid count");
        return -1;
    }

    err = hipMemPrefetchAsync(ptr, (size_t)count, dev);
    if (err != hipSuccess) {
        ndt_err_format(ctx, NDT_MemoryError,
            "hipMemPrefetchAsync: prefetching failed");
        return -1;
    }

    return 0;
}

extern "C" int
xnd_cuda_device_synchronize(ndt_context_t *ctx)
{
    hipError_t err;

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        ndt_err_format(ctx, NDT_RuntimeError,
            "cuda device synchronization failed");
        return -1;
    }

    return 0;
}
